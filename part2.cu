//
// Created by ayoumabr93 on 12/12/22.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cassert>


// déclaration des matrices
float raw_data[32*32];
float C1_data[6*28*28];
float S1_data[6*14*14];
float C1_kernel[6*5*5];

// fonction d'initialisation des matrices
void init_raw_data(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = rand() / (float)RAND_MAX;
    }
}

void init_C1_data(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = 0;
    }
}

void init_S1_data(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = 0;
    }
}

void init_C1_kernel(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = rand() / (float)RAND_MAX;
    }
}

void MatrixPrint(float *M, int n, int p){
    printf("[");
    for (int row=0; row<n; row++)
    {
        for(int col=0; col<p; col++)
        {
            if (row==n-1 & col==p-1){
                printf("%f]", M[row*p+col]);
            }else{
                printf("%f    ", M[row*p+col]);
            }
        }
        printf("\n");
    }
}

void conv2d(float *input, float *filter, int input_width, int input_height, int filter_width, int filter_height, float *output)
{
    int output_width = input_width - filter_width + 1;
    int output_height = input_height - filter_height + 1;

    // boucle pour parcourir chaque pixel de l'image d'entrée
    for (int i = 0; i < output_height; i++)
    {
        for (int j = 0; j < output_width; j++)
        {
            // initialisation de la valeur de sortie à 0
            output[i * output_width + j] = 0;

            // boucle pour appliquer le filtre à chaque pixel de l'image
            for (int k = 0; k < filter_height; k++)
            {
                for (int l = 0; l < filter_width; l++)
                {
                    // calcul de la convolution en multipliant chaque élément du filtre par la valeur correspondante de l'image d'entrée et en les additionnant
                    output[i * output_width + j] += filter[k * filter_width + l] * input[(i + k) * input_width + (j + l)];
                }
            }
        }
    }
}

void sub_sampling_2D(float* input, int rows, int cols, float* output) {
    // Vérifier que les tailles de l'entrée et de la sortie sont correctes
    assert(rows % 2 == 0 && cols % 2 == 0);

    // Échantillonner l'entrée pour générer la sortie
    for (int i = 0; i < rows; i += 2) {
        for (int j = 0; j < cols; j += 2) {
            output[(i / 2)*(cols/2)+j / 2] = (input[i * cols + j] + input[i * cols + j + 1] + input[(i + 1) * cols + j] + input[(i + 1) * cols + j + 1]) / 4.0;
        }
    }
}

__device__ float activation_tanh(float M) {
    return tanhf(M);
}


int main() {

    float raw_data[32*32];
    float C1_data[6*28*28];
    float S1_data[6*14*14];
    float C1_kernel[6*5*5];


    // initialisation des matrices
    init_raw_data(raw_data, 32*32);
    init_C1_data(C1_data, 6*28*28);
    init_S1_data(S1_data, 6*14*14);
    init_C1_kernel(C1_kernel, 6*5*5);

    // utilisation des matrices

    conv2d(raw_data, C1_kernel, 32, 32, 5, 5, C1_data);
    MatrixPrint(C1_data, 28,28);

    sub_sampling_2D(C1_data,28,28,S1_data);
    MatrixPrint(S1_data,14,14);
    return 0;
}
